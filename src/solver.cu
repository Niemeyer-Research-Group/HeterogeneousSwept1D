#include "hip/hip_runtime.h"
/**
    This file evaluates the Euler equations applied to the 1D Sod Shock Tube problem.  It demonstrates the numerical solution to this problem in parallel using the GPU. The solution procedure uses a second order finite volume scheme with a minmod limiter parameterized by the Pressure ratio at cells on a three point stencil.  The solution also uses a second-order in time (RK2 or midpoint) scheme.
    
    The problem may be evaluated in three ways: Classic, SharedGPU, and Hybrid.  Classic simply steps forward in time and calls the kernel once every timestep (predictor step or full step).  SharedGPU uses the GPU for all computation and applies the swept rule.  Hybrid applies the swept rule but computes the node on the boundary with the CPU.  
*/
/* 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

// Two primary strategies used in this code: global variables and templating using structures.

// Use: mpirun --bind-to-socket exe [args]

#include "decomposition/classicCore.h"
#include "decomposition/sweptCore.h"

#ifndef HDW
    #define HDW     #@WORKSTATION.json
#endif

/*
    TOOD
    - Swept always passes so what to do about bCond.
    - Make sure all struct variables are correctly initialized
    - Watch cluster video, try to run something like the Test bench
    - Write json for workstation situation.
    - Write the cluster explorer code.
    - Using an npm js solution for merging is a bad idea, try something else.
*/

// // This feels like a bad idea.
// void exitMerge()
// {
//     system("json-merge path/to/jsons/*.json")
// }

int main(int argc, char *argv[])
{   
    makeMPI(argc, &argv);

    // if (!ranks[1]) atexit(exitMerge);

    // Maybe using my new script and imporitng another json instead of this.

    // getDeviceInformation();

    // Maybe not declaring and mallocing right now.

    // Hardware information in json. Use hwloc, lspci, lstopo etc to get it.

    std::ifstream hwjson(HDW, std::ifstream::in);
    json hwJ;
    hwjson >> hwJ;
    hwjson.close();

    std::vector<int> gpuvec = hwJ["GPU"];
    std::vector<int smGpu(ivec.size())
    cGlob.nThreads = hwJ["nThreads"]; // Potetntial for non constant
    cGlob.hasGPU = ivec[ranks[1]];
    std::partial_sum(ivec.begin(), ivec.end(), smGpu.begin());
    cGlob.nGpu = smGpu.back();
    smGpu.insert(smGpu.begin(), 0);
    
    // Equation, grid, affinity data
    std::ifstream injson(argv[1], std::ifstream::in));
    json inJ;
    injson >> inJ;
    injson.close();

    parseArgs(inJ, argc, &argv);
    initArgs(inJ);

    /*  Essentially it should associate some unique (UUID?) for the GPU with the CPU. 
        Pretend you now have a (rank, gpu) map in all memory. because you could just retrieve it with a function.
    */


    states **state;
    double **xpts;
    if (cGlob.hasGpu)
    {
        **state = new state* [3];
        **xpts = new double* [3];
        // Now you have the index in smGpu[rank]*xg + xcp*rank  so get the k value with dx.

    }
    else
    {

    }

    int prt = 1 + 2*cGlob.hasGpu;
    

    


    int strt, npt, nar;


    for (int k=0; k<rank[2]; k++) 
    {

    }
         


    for (int k=0; k<; k++) 
        initialState(inJ, &state[k]->Q[0]);

    // We always know that there's some eqConsts struct that we need to 
    // to put into constant memory.
    // PROBABLY NEED TO CHECK TO MAKE SURE THERE"S A GPU FIRST.


    if (xgpu) hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), heqConsts, sizeof(eqConsts));

    int tstep = 1;
    // Start the counter and start the clock.
    MPI_Barrier(MPI_COMM_WORLD);
    hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the correct function with the correct algorithm.
    cout << scheme << " " ;
    double tfm;

    if (scheme)
    {
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, scheme-1, IC, T_final, freq, fwr);
    }
    else
    {
        tfm = classicWrapper(state, xpts, &tstep);
    }

    endMPI();

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timed, start, stop);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    std::ofstream soljson(argv[2]);
    soljson << solution;

    if (rank == 0)
    {
        //READ OUT JSONS
        
        timed *= 1.e3;

        double n_timesteps = tfm/dt;

        double per_ts = timed/n_timesteps;

        cout << n_timesteps << " timesteps" << endl;
        cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

        json timing;
        timing[dv][tpb][gpuA] = per_ts;

        std::ofstream timejson(argv[2]);
        timejson << std::setw(4) << timing << std::endl;
    }

    if (xgpu)
    {
        hipDeviceSynchronize();

        hipEventDestroy( start );
        hipEventDestroy( stop );
        hipDeviceReset();
    }

	return 0;
}