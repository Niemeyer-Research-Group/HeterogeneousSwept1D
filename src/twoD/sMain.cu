#include "hip/hip_runtime.h"
/*
    Entry point for hsweep.
*/
#include <fstream>

#include "heads.h"
#include "decomp.h"
#include "classic.h"
#include "swept.h"

/**
----------------------
    MAIN PART
----------------------
*/

int main(int argc, char *argv[])
{
    makeMPI(argc, argv);

    if (!ranks[1]) cudaRunCheck();

    #ifdef NOS
        if (!ranks[1]) std::cout << "No Solution Version." << std::endl;
    #endif

    std::string i_ext = ".json";
    std::string t_ext = ".csv";
    std::string myrank = std::to_string(ranks[1]);
    std::string scheme = argv[1];

    // Equation, grid, affinity data
    std::ifstream injson(argv[2], std::ifstream::in);
    injson >> inJ;
    injson.close();

    parseArgs(argc, argv);
    initArgs();

    //OH LORD

    int exSpace = (!scheme.compare("S") ? cGlob.ht : 2);
    states *state;

    int nodeAlloc = (cGlob.nodeSidex * cGlob.blockSide + exSpace) *  (cGlob.nodeSidey * cGlob.blockSide + exSpace);
    int gpuAlloc = cGlob.nodeSidex * cGlob.blockSide * cGlob.gpux + exSpace) *  (cGlob.nodeSidey * cGlob.blockSide * cGlob.gpuy + exSpace);

    if (cGlob.hasGpu)
    {
        hipHostAlloc((void **) &state, gpuAlloc * cGlob.szState, hipHostMallocDefault);
    }
    else
    {
        state = (states*) malloc(nodeAlloc * cGlob.szState);
    }

    // NOW WE MUST ASSIGN PARTICULAR AREAS FOR EACH NODE, INITIALIZE THE ARRAY AND THE TWO ARRAYS OF POINTERS TO PARTS OF THE ARRAY.


    int prevGpu=0; //Get the number of GPUs in front of the current process.
    int gpuPlaces[nprocs]; //Array of 1 or 0 for number of GPUs assigned to process

    //If there are no GPUs or if the GPU Affinity is 0, this block is unnecessary.
    if (cGlob.nGpu > 0)
    {
        MPI_Allgather(&cGlob.hasGpu, 1, MPI_INT, &gpuPlaces[0], 1, MPI_INT, MPI_COMM_WORLD);
        for (int k=0; k<ranks[1]; k++) prevGpu+=gpuPlaces[k];
    }

    cGlob.xStart = cGlob.xcpu * ranks[1] + cGlob.xg * prevGpu;
    states **state;

    std::string pth = string(argv[3]);

    writeOut(state, 0.0);

    // If you have selected scheme I, it will only initialize and output the initial values.

    if (scheme.compare("I"))
    {
        int tstep = 1;
        double timed, tfm;

		if (!ranks[1])
		{
            printf ("Scheme: %s - Grid Size: %d - Affinity: %.2f\n", scheme.c_str(), cGlob.nX, cGlob.gpuA);
            printf ("threads/blk: %d - timesteps: %.2f\n", cGlob.tpb, cGlob.tf/cGlob.dt);
		}

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = MPI_Wtime();

        if (!scheme.compare("C"))
        {
            tfm = classicWrapper(state, &tstep);
        }
        else if  (!scheme.compare("S"))
        {
            tfm = sweptWrapper(state, &tstep);
        }
        else
        {
            std::cerr << "Incorrect or no scheme given" << std::endl;
        }

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = (MPI_Wtime() - timed);
        if (cGlob.hasGpu)  
		{
			hipError_t error = hipGetLastError();
        	if(error != hipSuccess)
        	{
            	// print the CUDA error message and exit
            	printf("CUDA error: %s\n", hipGetErrorString(error));
            	exit(-1);
        	}
			hipDeviceSynchronize();
		}

        writeOut(state, tfm);

        if (!ranks[1])
        {
            timed *= 1.e6;

            double n_timesteps = tfm/cGlob.dt;

            double per_ts = timed/n_timesteps;

            std::cout << n_timesteps << " timesteps" << std::endl;
            std::cout << "Averaged " << per_ts << " microseconds (us) per timestep" << std::endl;

            // Write out performance data as csv
            std::string tpath = pth + "/t" + fspec + scheme + t_ext;
            FILE * timeOut;
            timeOut = fopen(tpath.c_str(), "a+");
            fseek(timeOut, 0, SEEK_END);
            int ft = ftell(timeOut);
            if (!ft) fprintf(timeOut, "tpb,gpuA,nX,time\n");
            fprintf(timeOut, "%d,%.4f,%d,%.8f\n", cGlob.tpb, cGlob.gpuA, cGlob.nX, per_ts);
            fclose(timeOut);
        }
    }
        //WRITE OUT JSON solution to differential equation

	#ifndef NOS
        std::string spath = pth + "/s" + fspec + "_" + myrank + i_ext;
        std::ofstream soljson(spath.c_str(), std::ofstream::trunc);
        if (!ranks[1]) solution["meta"] = inJ;
        soljson << solution;
        soljson.close();
	#endif

    if (cGlob.hasGpu)
    {
        hipHostFree(state);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
    else
    {
        free(state);
    }

    endMPI();
    return 0;
}

