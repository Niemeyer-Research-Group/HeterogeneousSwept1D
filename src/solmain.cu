#include "hip/hip_runtime.h"
// ALLONE

// Well we definitely need to get rid of the xpts.  Really I need to concentrate on getting the output right so I can check the answers.  Then, if they're right, we can worry about streamlining this. Partly main problem, the keys in the output json are strings. Could read each in and then make it a data frame from dict.

#include <fstream>

#define cudaCheckError(ans) { cudaCheck((ans), __FILE__, __LINE__); }
inline void cudaCheck(hipError_t code, const char *file, int line, bool abort=false) 
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#include "heads.h"
#include "decomp.h"
#include "classic.h"
#include "swept.h"

/**
----------------------
    MAIN PART
----------------------
*/

#ifndef HDW
    #define HDW     "hardware/WORKSTATION.json"
#endif

std::vector<int> jsonP(jsons jp, size_t sz)
{
	std::vector <int> outv;
	for(int i=0; i<sz; i++)
	{
		outv.push_back(jp[i].asInt());
	}
	return outv;
}

int main(int argc, char *argv[])
{
    makeMPI(argc, argv);

    std::string ext = ".json";
    std::string myrank = std::to_string(ranks[1]);
    std::string sout = argv[3];
    sout.append(myrank);
    sout.append(ext); 
    std::string scheme = argv[1];

    std::ifstream hwjson(HDW, std::ifstream::in);
    jsons hwJ;
    hwjson >> hwJ;
    hwjson.close();

    // However you coe by these vectors are somewhat immaterial.  Could do a test on the cluster before running.
    cGlob.nGpu = hwJ["nGpu"].asInt();
    std::vector<int> gpuvec = jsonP(hwJ["pGpu"], cGlob.nGpu);
    std::vector<int> myGPU = jsonP(hwJ["gpuID"], cGlob.nGpu);
    int gpuID = -1;
    for (int k=0; k<cGlob.nGpu; k++)
    {
        if (ranks[1] == gpuvec[k])
        {
            cGlob.hasGpu = true;
            gpuID = myGPU[k];
        }
    }
    int smGpu = std::count_if(gpuvec.begin(), gpuvec.end(), [](int i){return i<ranks[1] == 1;});

    // Equation, grid, affinity data
    std::ifstream injson(argv[2], std::ifstream::in);
    injson >> inJ;
    injson.close();

    parseArgs(argc, argv);
    initArgs();

    /*
        Essentially it should associate some unique (UUID?) for the GPU with the CPU. 
        Pretend you now have a (rank, gpu) map in all memory. because you could just retrieve it with a function.
    */
    int strt = cGlob.xcpu * ranks[1] + cGlob.xg * smGpu; 
    states **state;

    int exSpace = ((int)!scheme.compare("S") * cGlob.ht) + 2;
    int xc = (cGlob.hasGpu) ? cGlob.xcpu/2 : cGlob.xcpu;
    int nrows = (cGlob.hasGpu) ? 3 : 1;
    int xalloc = xc + exSpace;

    std::string pth = string(argv[3]);
    std::vector<int> xpts(1, strt); //
    std::vector<int> alen(1, xc);
    if(!ranks[1]) std::cout << "Before initial values." << std::endl;
    if (cGlob.hasGpu)
    {
        hipSetDevice(gpuID);

        state = new states* [3];
        cudaCheckError(hipHostAlloc((void **) &state[0], xalloc * cGlob.szState, hipHostMallocDefault));
        cudaCheckError(hipHostAlloc((void **) &state[1], (cGlob.xg + exSpace) * cGlob.szState, hipHostMallocDefault));
        cudaCheckError(hipHostAlloc((void **) &state[2], xalloc * cGlob.szState, hipHostMallocDefault));

        xpts.push_back(strt + xc);
        alen.push_back(cGlob.xg + exSpace);
        xpts.push_back(strt + xc + cGlob.xg);
        alen.push_back(xc);

        cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), &heqConsts, sizeof(eqConsts)));

        if (sizeof(REAL)>6)
        {
            cudaCheckError(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
        }
    }
    else
    {
        state = new states* [1];
        cudaCheckError(hipHostAlloc((void **) &state[0], xalloc * cGlob.szState, hipHostMallocDefault));
    }

    for (int i=0; i<nrows; i++)
    {
	std::cout << ranks[1] << " " << i << " " << alen[i] << " " << nrows << " " << xpts[i] << " " << " " << exSpace << " " << cGlob.xg << std::endl;
        for (int k=0; k<alen[i]; k++)  initialState(inJ, state[i], k, xpts[i]);
        for (int k=1; k<=alen[i]; k++)  solutionOutput(state[i], 0.0, k, xpts[i]);
    }

    // If you have selected scheme I, it will only initialize and output the initial values.
    if (!ranks[1])	std::cout << "Initial Values Instantiated." << std::endl;

    if (scheme.compare("I"))
    {
        int tstep = 1;
        double timed, tfm;

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = MPI_Wtime();

        if (!scheme.compare("C"))
        {
            tfm = classicWrapper(state, xpts, alen, &tstep);
        }
        else if  (!scheme.compare("S"))
        {
            tfm = sweptWrapper(state, xpts, alen, &tstep);
        }
        else
        {
            std::cerr << "Incorrect or no scheme given" << std::endl;
        }

        if (cGlob.hasGpu)
        {
            hipDeviceSynchronize();
        }

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = (MPI_Wtime() - timed);

        for (int i=0; i<nrows; i++)
        {
            for (int k=1; k<=alen[i]; k++)  solutionOutput(state[i], tfm, k, xpts[i]);
        }

        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            // print the CUDA error message and exit
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        if (!ranks[1])
        {
            //READ OUT JSONS
            timed *= 1.e6;

            double n_timesteps = tfm/cGlob.dt;

            double per_ts = timed/n_timesteps;

            std::cout << n_timesteps << " timesteps" << std::endl;
            std::cout << "Averaged " << per_ts << " microseconds (us) per timestep" << std::endl;

            std::string tpath = pth + "/t" + fspec + scheme + ext;
            try {
                std::ifstream tjson(tpath, std::ifstream::in);
                tjson >> timing;
                tjson.close();
            }
            catch (...) {}

            std::string tpbs = std::to_string(cGlob.tpb);
            std::string nXs = std::to_string(cGlob.nX);
            std::string gpuAs = std::to_string(cGlob.gpuA);
            std::cout << cGlob.gpuA << std::endl;

            
            std::ofstream timejson(tpath.c_str(), std::ofstream::trunc);
            timing[tpbs][nXs][gpuAs] = per_ts;
            timejson << timing;
            timejson.close();
        }
    }

    std::string spath = pth + "/s" + fspec + "_" + std::to_string(ranks[1]) + ext;
    std::ofstream soljson(spath.c_str(), std::ofstream::trunc);
    if (!ranks[1]) solution["meta"] = inJ;
    soljson << solution;
    soljson.close();

    endMPI();

    for (int k=0; k<nrows; k++)
    {
        hipHostFree(state[k]);
    }
    delete[] state;   
    if (cGlob.hasGpu)
    {
        hipDeviceSynchronize();
        hipDeviceReset();
    }
    return 0;
}
