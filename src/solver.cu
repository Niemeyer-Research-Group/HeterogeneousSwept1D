#include "hip/hip_runtime.h"
/**
    This file evaluates the Euler equations applied to the 1D Sod Shock Tube problem.  It demonstrates the numerical solution to this problem in parallel using the GPU. The solution procedure uses a second order finite volume scheme with a minmod limiter parameterized by the Pressure ratio at cells on a three point stencil.  The solution also uses a second-order in time (RK2 or midpoint) scheme.
    
    The problem may be evaluated in three ways: Classic, SharedGPU, and Hybrid.  Classic simply steps forward in time and calls the kernel once every timestep (predictor step or full step).  SharedGPU uses the GPU for all computation and applies the swept rule.  Hybrid applies the swept rule but computes the node on the boundary with the CPU.  
*/
/* 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

// Two primary strategies used in this code: global variables and templating using structures.

#include "decomposition/classicCore.h"
#include "decomposition/sweptCore.h"

int main( int argc, char *argv[] )
{   
    // Set shared memory banks to double if REAL is double.
    if (sizeof(REAL)>6) hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    states *state;
    double *xpts;

    std::ifstream injson(argv[1]);
    json inJ;
    injson >> inJ;

    makeMPI(int argc, char* argv[]);

    hBound[0] = {};
    hBound[1] = {};

    for (int k=0; k<dv; k++) initialState(inJ, &state[k]->Q[0]);

    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const double dt = atof(argv[3]);
	const double tf = atof(argv[4]) - QUARTER*dt; //Finish time
    const double freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //2 for Alternate, 1 for GPUShared, 0 for Classic
    const int bks = dv/tpb; //The number of blocks
    const double dx = lx/((REAL)dv-TWO);
    char const *prec;
    prec = (sizeof(REAL)<6) ? "Single": "Double";

    dimz.dt_dx = dt/dx; // dt/dx
    dimz.base = tpb+4;
    dimz.idxend = dv-1;
    dimz.idxend_1 = dv-2;

    for (int k=-2; k<3; k++) dimz.hts[k+2] = (tpb/2) + k;
        
    eCheckIn(dv, tpb, argc); //Initial error checking.

    // We always know that there's some eqConsts struct that we need to 
    // to put into constant memory.
    // PROBABLY NEED TO CHECK TO MAKE SURE THERE"S A GPU FIRST.
	hipMemcpyToSymbol(HIP_SYMBOL(deqConsts),&heqConsts,sizeof(eqConsts));

    if (dimz.dt_dx > .21)
    {
        cout << "The value of dt/dx (" << dimz.dt_dx << ") is too high.  In general it must be <=.21 for stability." << endl;
        exit(-1);
    }

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[7],ios::trunc);
    fwr.precision(10);

    // Start the counter and start the clock.
    hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the correct function with the correct algorithm.
    cout << scheme << " " ;
    double tfm;
    if (scheme)
    {
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, scheme-1, IC, T_final, freq, fwr);
    }
    else
    {
        tfm = classicWrapper(bks, tpb, dv, dt, tf, IC, T_final, freq, fwr);
    }

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    timed *= 1.e3;

    double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>7)
    {
        ofstream ftime;
        ftime.open(argv[8],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

    hipDeviceSynchronize();

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();

	return 0;
}