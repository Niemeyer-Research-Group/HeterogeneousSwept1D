#include "hip/hip_runtime.h"
// ALLONE

// Well we definitely need to get rid of the xpts.  Really I need to concentrate on getting the output right so I can check the answers.  Then, if they're right, we can worry about streamlining this. Partly main problem, the keys in the output json are strings. Could read each in and then make it a data frame from dict.

#include <fstream>

#define cudaCheckError(ans) { cudaCheck((ans), __FILE__, __LINE__); }
inline void cudaCheck(hipError_t code, const char *file, int line, bool abort=false) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#include "heads.h"
#include "decomp.h"
#include "classic.h"
#include "swept.h"
#include <iomanip>

/**
----------------------
    MAIN PART
----------------------
*/

#ifndef HDW
    #define HDW     "WORKSTATION.json"
#endif

std::vector<int> jsonP(jsons jp, size_t sz)
{
	std::vector <int> outv;
	for(int i=0; i<sz; i++)
	{
		outv.push_back(jp[i].asInt());
	}
	return outv;
}

int main(int argc, char *argv[])
{   
    makeMPI(argc, argv);

    std::string ext = ".json";
    std::string myrank = std::to_string(ranks[1]);
    std::string sout = argv[3];
    sout.append(myrank);
    sout.append(ext); 
    std::string scheme = argv[1];

    std::ifstream hwjson(HDW, std::ifstream::in);
    jsons hwJ;
    hwjson >> hwJ;
    hwjson.close();

    std::vector<int> gpuvec = jsonP(hwJ["GPU"], 1);
    std::vector<int> smGpu(gpuvec.size());
    cGlob.hasGpu = gpuvec[ranks[1]];
    std::partial_sum(gpuvec.begin(), gpuvec.end(), smGpu.begin());
    cGlob.nGpu = smGpu.back();
    smGpu.insert(smGpu.begin(), 0);
    std::vector <int> myGPU = jsonP(hwJ["gpuID"], 1);
    int gpuID = myGPU[ranks[1]];
    
    // Equation, grid, affinity data
    std::ifstream injson(argv[2], std::ifstream::in);
    injson >> inJ;
    injson.close();

    parseArgs(argc, argv);
    initArgs();

    /*  
        Essentially it should associate some unique (UUID?) for the GPU with the CPU. 
        Pretend you now have a (rank, gpu) map in all memory. because you could just retrieve it with a function.
    */

    // Well we definitely need to get rid of the xpts.  Really I need to concentrate on getting the output right so I can check the answers.  Then, if they're right, we can worry about streamlining this. 
    int strt = cGlob.xcpu * ranks[1] + cGlob.xg * cGlob.hasGpu * smGpu[ranks[1]]; 
    states **state;

    int exSpace = ((int)!scheme.compare("S") * cGlob.ht) + 2;
    int xc = (cGlob.hasGpu) ? cGlob.xcpu/2 : cGlob.xcpu;
    int nrows = (cGlob.hasGpu) ? 3 : 1;
    int xalloc = xc + exSpace;

    std::string pth = string(argv[3]);
    std::vector<int> xpts(strt); //
    std::vector<int> alen(xc);

    if (cGlob.hasGpu)
    {
        hipSetDevice(gpuID);
        
        state = new states* [3];
        for 
        cudaCheckError(hipHostAlloc((void **) &state[0], xalloc * cGlob.szState, hipHostMallocDefault));
        cudaCheckError(cudaCheckError(hipHostAlloc((void **) &state[1], (cGlob.xg + exSpace) * cGlob.szState, hipHostMallocDefault));
        hipHostAlloc((void **) &state[2], xalloc * cGlob.szState, hipHostMallocDefault));

        xpts.push_back(strt + xc);
        alen.push_back(cGlob.xg)
        xpts.push_back(strt + xc + cGlob.xg);
        alen.push_back(xalloc);

        hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), &heqConsts, sizeof(eqConsts));

        if (sizeof(REAL)>6) 
        {
            cudaCheckError(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte)0;
        }
    }
    else 
    {
        state = new states* [1];
        cudaCheckError(hipHostAlloc((void **) &state[0], xalloc * cGlob.szState, hipHostMallocDefault));   
    }

    for (int i=0; i<nrows; i++)
    {
        for (int k=0; k<alen[i] + exSpace; k++)  initialState(inJ, state[i], k, xpts[i]);
        for (int k=1; k<=alen[i]; k++)  solutionOutput(state[i], 0.0, k, xpts[i]); 

    }

    // Check CUDA alloc.
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // If you have selected scheme I, it will only initialize and output the initial values.
    if (scheme.compare("I"))
    {
        int tstep = 1;
        double timed, tfm;

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = MPI_Wtime();
        cout << "Made it to Calling the function " << endl;

        if (!scheme.compare("C"))
        {
            tfm = classicWrapper(state, xpts, alen, &tstep);
        }
        else if  (!scheme.compare("S"))
        {
            tfm = sweptWrapper(state, xpts, alen, &tstep);
        }
        else
        {
            std::cerr << "Incorrect or no scheme given" << std::endl;
        }

        if (cGlob.hasGpu)
        {
            hipDeviceSynchronize();
        }

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = (MPI_Wtime() - timed);

        for (int i=0; i<nrows; i++)
        {
            for (int k=1; k<=alen[i]; k++)  solutionOutput(state[i], 0.0, k, xpts[i]);
        }  

        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            // print the CUDA error message and exit
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        if (!ranks[1])
        {
            //READ OUT JSONS
            
            timed *= 1.e6;

            double n_timesteps = tfm/cGlob.dt;

            double per_ts = timed/n_timesteps;

            std::cout << n_timesteps << " timesteps" << std::endl;
            std::cout << "Averaged " << per_ts << " microseconds (us) per timestep" << std::endl;

            // Equation, grid, affinity data
            try {
                std::ifstream tjson(argv[4], std::ifstream::in);
                tjson >> timing;
                tjson.close();
            }
            catch (...) {}

            std::string tpbs = std::to_string(cGlob.tpb);
            std::string nXs = std::to_string(cGlob.nX);
            std::string gpuAs = std::to_string(cGlob.gpuA);
            std::cout << cGlob.gpuA << std::endl;

            std::string spath = pth + "/t" + fspec + ext;
            std::ofstream timejson(spath.c_str(), std::ofstream::trunc);
            timing[tpbs][nXs][gpuAs] = per_ts;
            timejson << timing;
            timejson.close();
        }
    }

    std::string spath = pth + "/s" + fspec + "_" + std::to_string(ranks[1]) + ext;
    std::ofstream soljson(spath.c_str(), std::ofstream::trunc);
    if (!ranks[1]) solution["meta"] = inJ;
    soljson << solution;
    soljson.close();

    endMPI();

    if (cGlob.hasGpu)
    {
        hipDeviceSynchronize();

        for (int k=0; k<3; k++)
        {
            hipHostFree(xpts[k]);
            hipHostFree(state[k]);
        }
        
        delete[] xpts;
        delete[] state;
        hipDeviceReset();
    }
    else
    {
        hipHostFree(xpts[0]);
        hipHostFree(state[0]);
        delete[] xpts;
        delete[] state;
    }
    return 0;

}
