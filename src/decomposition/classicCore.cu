#include "hip/hip_runtime.h"
/**
    The Classic Functions for the stencil operation
*/

// Perhaps http://www.cplusplus.com/reference/unordered_map/unordered_map/insert/
// For json.

#include "classicCore.h"

/** 
    Classic kernel for simple decomposition of spatial domain.

    @param States The working array result of the kernel call before last (or initial condition) used to calculate the RHS of the discretization
    @param finalstep Flag for whether this is the final (True) or predictor (False) step
*/
__global__ void classicStep(states *state, int ts)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x + 1; //Global Thread ID (one extra)

    stepUpdate(state, gid, ts)
}

void classicStepCPU(states *state, int numx, int tstep)
{
    for (int k=1; k<numx; k++)
    {
        stepUpdate(state, k, tstep)
    }
}

void classicPassLeft(states *state, int idxend, int tstep))
{   
    if (bCond[0])
    {
        MPI_Isend(&state[1], 1, struct_type, ranks[0], TAGS(tstep),
                MPI_COMM_WORLD, &req[0]);

        MPI_recv(&state[0], 1, struct_type, ranks[0], TAGS(tstep+100), 
                MPI_COMM_WORLD, MPI_STATUS_IGNORE); 
    }
                                     
}

void classicPassRight(states *state, int idxend)
{
    if (bCond[1]) 
    {
        MPI_Isend(&state[idxend-1], 1, struct_type, ranks[2], TAGS(tstep+100),
                MPI_COMM_WORLD, &req[1]);

        MPI_recv(&state[idxend], 1, struct_type, ranks[2], TAGS(tstep), 
                MPI_COMM_WORLD,  MPI_STATUS_IGNORE);
    }
}

// We are working with the assumption that the parallelism is too fine to see any benefit.
// Still struggling with the idea of the local vs parameter arrays.
// Classic Discretization wrapper.
double classicWrapper(states **state, double **xpts, int *tstep)
{
    cout << "Classic Decomposition" << endl;

    double t_eq = 0.0;
    double twrite = cGlob.freq - QUARTER*cGlob.dt;

    if (cGlob.hasGpu) // If there's no gpu assigned to the process this is 0.
    {
        const int xc = cGlob.xcpu/2, xcp = xc+1, xcpp = xc+2;
        const int xgp = cGlob.xg+1, xgpp = cGlob.xg+2;
        const int gpusize =  cGlob.szState * xgpp;
        const int cpuzise = cGlob.szState * xcpp;

        states *dState;
        
        hipMalloc((void **)&dState, gpusize;
        // Copy the initial conditions to the device array.
        hipMemcpy(dState, state[1], gpusize, hipMemcpyHostToDevice);

        // Four streams for four transfers to and from cpu.
        hipStream_t st1, st2, st3, st4;
        hipStreamCreate(&st1);
        hipStreamCreate(&st2);
        hipStreamCreate(&st3);
        hipStreamCreate(&st4);

        while (t_eq < cGlob.tf)
        {
            classicDecomp <<< cGlob.bks, cGlob.tpb >>> (dState, tstep);

            #pragma omp parallel sections num_threads(2)
            {
                #pragma omp section
                {
                    classicStepCPU(state[0], xcp, tstep);
                }
                #pragma omp section
                {
                    classicStepCPU(state[2], xcp, tstep);
                }
            }
            
            // Host to device first.
            # pragma omp parallel sections num_threads(3)
            {
                #pragma omp section
                {
                    hipMemcpyAsync(dState, state[0] + xc, cGlob.szState, hipMemcpyHostToDevice, st1);
                    hipMemcpyAsync(dState + xgp, state[2] + 1, cGlob.szState, hipMemcpyHostToDevice, st2);
                    hipMemcpyAsync(state[0] + xcp, dState + 1, cGlob.szState, hipMemcpyDeviceToHost, st3);
                    hipMemcpyAsync(state[0, dState + cGlob.xg, cGlob.szState, hipMemcpyDeviceToHost, st4);
                }
                #pragma omp section
                {
                    classicPassRight(state[2], xcp);
                }
                #pragma omp section
                {
                    classicPassLeft(state[0], xcp);
                }
            }
            
            // Increment Counter and timestep
            if (MODULA(tstep)) t_eq += dt;
            tstep++

            if (t_eq > twrite)
            {
                hipMemcpy(state[1], dState, gpubytes, hipMemcpyDeviceToHost);

                for (int k=1; k<xcp; k++) solutionOutput(state[0]+k, xpts[0][k], t_eq);                
                for (int k=1; k<xcp; k++) solutionOutput(state[2]+k, xpts[2][k], t_eq);
                for (int k=1; k<xgp; k++) solutionOutput(state[1]+k, xpts[1][k], t_eq);

                twrite += freq;
            }
        }

        hipMemcpy(state[1], dState, gpubytes, hipMemcpyDeviceToHost);

        hipStreamDestroy(st1);
        hipStreamDestroy(st2);
        hipStreamDestroy(st3);
        hipStreamDestroy(st4);
        
        for (int k=1; k<xcp; k++) solutionOutput(state[0]+k, xpts[0][k], t_eq);
        for (int k=1; k<xcp; k++) solutionOutput(state[2]+k, xpts[2][k], t_eq);
        for (int k=1; k<xgp; k++) solutionOutput(state[1]+k, xpts[1][k], t_eq);

        hipFree(dState);
    }
    else
    {
        while (t_eq < cGlob.tf)
        {
            // Alias the pointer to make it clearer.
            states *cState = state[0];
            int xcp = cGlob.xcpu + 1;

            classicStepCPU(cState, xcp);
            if (MODULA(tstep)) t_eq += cGlob.dt;
            tstep++

            #pragma omp parallel sections num_threads(2)
            {
                #pragma omp section
                {
                    classicPassRight(cState, xcp);
                }
                #pragma omp section
                {
                    classicPassLeft(cState, xcp);
                }
            }

            if (t_eq > twrite)
            
                for (int k=1; k<xcp; k++) solutionOutput(cState[k], xpts[0][k], t_eq);
                twrite += cGlob.freq;
            }

        for (int k=1; k<xcp; k++) solutionOutput(cState[k], xpts[0][k], t_eq);
    }
    return t_eq;
}
