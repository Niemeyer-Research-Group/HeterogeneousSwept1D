/**
    Functions that enable the decomposition that are the same 
    for classic and swept versions.
*/

#include "decompCore.h"

//Always prepared for periodic boundary conditions.
void makeMPI(int argc, char* argv[])
{
    mpi_type(&struct_type);
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &ranks[1]);
	MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    lastproc = nprocs-1;
	ranks[0] = (ranks[1]-1) % nprocs;
    ranks[2] = (ranks[1]+1) % nprocs;
}

void getDeviceInformation();
{
    hipGetDeviceCount(nGpu);

    if (nGpu)
    {
        cudaGetDeviceProp(&props);
    }
    
    nthreads = omp_get_num_procs();

    // From this I want what GPUs each proc can see, and how many threads they can make
    // This may require nvml to get the UUID of the GPUS, pass them all up to the 
    // Master proc to decide which proc gets which gpu.
}

void delegateDomain()
{
    // Set shared memory banks to double if REAL is double.
    if (sizeof(REAL)>6 && xgpu) 
    {
        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    }
}

/* 
    Takes any extra command line arguments which override json default args and inserts 
    them into the json type which will be read into variables in the next step.

    Arguments are key, value pairs all lowercase keys, no dash in front of arg.
*/
void parseArgs(json inJ, int argc, char *argv[]);
{
    if (argc>6)
    {
        for (int k=6; k<argc; k+=2)
        {
            inJ[argv[k]] = argv[k+1];
        }
    }
}

void initArgs(json inJ);
{

    cGlob.lx = inJ["lx"]
    cGlob.szState = sizeof(states);
    cGlob.base = cGlob.tpb+2;
    cGlob.tpbp = cGlob.tpb+1;
    cGlob.ht = cGlob.tpb/2;
    cGlob.htm = cGlob.ht-1;
    cGlob.tpb = inJ["tpb"];
    cGlob.gpuA = inJ["gpuA"];
    cGlob.dt = inJ["dt"];
    cGlob.tf = inJ["tf"];
    cGlob.freq = inJ["freq"];
    cGlob.nX = inJ["nX"];

    cGlob.xg = ((cGlob.tpb * cGlob.gpuA)/32) * 32;  // Number of gpu spatial points.
    cGlob.xcpu = cGlob.nThreads * cGlob.tpb;
    cGlob.xWave = (nprocs * cGlob.xcpu + cGlob.nGpu * cGlob.xg); // Number of points on a device x number of devices.
    cGlob.nWaves = CEIL(cGlob.xWave, cGlob.nX);
    cGlob.nX = cGlob.nWaves*cGlob.xWave; // Now it's an even wave of spatial points.

    cGlob.tpbp = cGlob.tpb + 1;
    cGlob.base = cGlob.tpb + 2;
    cGlob.ht = cGlob.tpb/2;
    cGlob.htm = cGlob.ht - 1;
    cGlob.htp = cGlob.ht + 1;

    cGlob.dx = cGlob.lx/(cGlob.nX - 2.0); // Spatial step
    inJ["dx"] = cGlob.dx; // To send back to equation folder.  It aay need it, it may not.

    equationSpecificArgs(json inJ); 

    // Maybe here assign gpus
    // return bool
    // hasGpu = gpuAssign();

    cGlob.xg = cGlob.xg * cGlob.hasGpu;
    cGlob.xtot = cGlob.xcpu + cGlob.xg + cGlob.hasGpu*4 + 2; // cpu points + gpu points + inner ghosts + outer ghosts

    
    // Swept Always Passes!
    enum
    {
        // If BCTYPE == "Dirichlet"
        if (!ranks[1]) cGlob.bCond[0] = false;
        if (ranks[1] == lastproc) cGlob.bCond[1] = false;
        // If BCTYPE == "Periodic"
            // Don't do anything.
    }


}
// THIS IS GREAT BUT YOU CAN'T PASS IT BACK BECAUSE TYPES!
// Maybe.
void solutionOutput(REALthree outState, REAL tstamp, REAL xpt)
{
    for (int k=0; k<NVARS; k++)
    {
        solution[outVars[k]][tstamp][xpt] = printout(k, outVec); 
    }
}

void endMPI()
{
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();
}

