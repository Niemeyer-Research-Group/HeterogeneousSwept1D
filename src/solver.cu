#include "hip/hip_runtime.h"
/**
    This file evaluates the Euler equations applied to the 1D Sod Shock Tube problem.  It demonstrates the numerical solution to this problem in parallel using the GPU. The solution procedure uses a second order finite volume scheme with a minmod limiter parameterized by the Pressure ratio at cells on a three point stencil.  The solution also uses a second-order in time (RK2 or midpoint) scheme.
    
    The problem may be evaluated in three ways: Classic, SharedGPU, and Hybrid.  Classic simply steps forward in time and calls the kernel once every timestep (predictor step or full step).  SharedGPU uses the GPU for all computation and applies the swept rule.  Hybrid applies the swept rule but computes the node on the boundary with the CPU.  
*/
/* 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

// Two primary strategies used in this code: global variables and templating using structures.

// Use: mpirun --bind-to-socket exe [args]

#include "decomposition/classicCore.h"
#include "decomposition/sweptCore.h"

#ifndef HDW
    #define HDW     #@WORKSTATION.json
#endif

/*
    TOOD
    - Swept always passes so what to do about bCond.
    - Make sure all struct variables are correctly initialized
    - Watch cluster video, try to run something like the Test bench
    - Write json for workstation situation.
    - Write the cluster explorer code.
    - Using an npm js solution for merging is a bad idea, try something else.
*/

// // This feels like a bad idea.
// void exitMerge()
// {
//     system("json-merge path/to/jsons/*.json")
// }

int main(int argc, char *argv[])
{   
    makeMPI(argc, &argv);

    std::string ext = ".json";
    std::string myrank = ranks[1].to_string();
    std::string sout = argv[3];
    sout.append(myrank);
    sout.append(ext); 
    int scheme;

    if (!argv[1].compare("C"))
    {

    }
    else if  (!argv[1].compare("C"))
    // if (!ranks[1]) atexit(exitMerge);

    std::ifstream hwjson(HDW, std::ifstream::in);
    json hwJ;
    hwjson >> hwJ;
    hwjson.close();

    std::vector<int> gpuvec = hwJ["GPU"];
    std::vector<int> smGpu(ivec.size());
    cGlob.nThreads = hwJ["nThreads"]; // Potetntial for non constant
    cGlob.hasGPU = ivec[ranks[1]];
    std::partial_sum(ivec.begin(), ivec.end(), smGpu.begin());
    cGlob.nGpu = smGpu.back();
    smGpu.insert(smGpu.begin(), 0);
    int gpuID = hwJ["gpuID"];
    
    // Equation, grid, affinity data
    std::ifstream injson(argv[1], std::ifstream::in));
    json inJ;
    injson >> inJ;
    injson.close();

    parseArgs(inJ, argc, &argv);
    initArgs(inJ);

    /*  
        Essentially it should associate some unique (UUID?) for the GPU with the CPU. 
        Pretend you now have a (rank, gpu) map in all memory. because you could just retrieve it with a function.
    */

    int strt = cGlob.xcpu * ranks[1] + cGlob.xg * cGlob.hasGpu * smGput[ranks[1]]; //
    states **state;
    double **xpts;

    int exSpace = (cGlob.hasGpu) ? cGlob.htp : 2;
    int xc = (cGlob.hasGpu) ? cGlob.xcpu/2 : cGlob.xcpu;
    int xalloc = xc + exSpace;

    if (cGlob.hasGpu)
    {
        hipSetDevice(gpuID);
        
        **state = new state* [3];
        **xpts = new double* [3];
        hipHostAlloc((void **) &xpts[0], xc * sizeof(double), hipHostMallocDefault);
        hipHostAlloc((void **) &xpts[1], (cGlob.xg + exSpace) * sizeof(double), hipHostMallocDefault);
        hipHostAlloc((void **) &xpts[2], xc * sizeof(double), hipHostMallocDefault);
        hipHostAlloc((void **) state[0], xalloc * cGlob.szState, hipHostMallocDefault);
        hipHostAlloc((void **) state[1], (cGlob.xg + exSpace) * cGlob.szState, hipHostMallocDefault);
        hipHostAlloc((void **) state[2], xalloc * cGlob.szState, hipHostMallocDefault);

        int pone = (strt + xc) 
        int ptwo = (pone + cGlob.xg);

        for (int k=1; k<=xc; k++) 
        {
            initialState(inJ, k, strt, state[0], xpts[0]); 
            initialState(inJ, k, ptwo, state[2], xpts[2]); 
        }

        for (int k=1; k <= cGlob.xg; k++)  initialState(inJ, k, pone, state[1], xpts[1]); 

        // Now you have the index in smGpu[rank]*xg + xcp*rank  so get the k value with dx.
        hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), heqConsts, sizeof(eqConsts));
    }
    else
    {
        **state = new state* [1];
        **xpts = new double* [1];
        hipHostAlloc((void **) xpts[0], xalloc * sizeof(double), hipHostMallocDefault);
        hipHostAlloc((void **) state[0], xalloc * cGlob.szState, hipHostMallocDefault);
        for (int k=1; k<cGlob.xc); k++)  initialState(inJ, k, strt, state[0], xpts[0]); 
    }
    int tstep = 1;
    // Start the counter and start the clock.
    MPI_Barrier(MPI_COMM_WORLD);
    hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the correct function with the correct algorithm.
    double tfm;

    if (!argv[1].compare("C"))
    {
        tfm = classicWrapper(state, xpts, &tstep);
    }
    else if  (!argv[1].compare("S"))
    {
        tfm = sweptWrapper(state, xpts, &tstep);
    }
    else
    {
        std::cerr << "Incorrect or no scheme given" << std::endl;
    }

    MPI_Barrier(MPI_COMM_WORLD);

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timed, start, stop);

    endMPI();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    std::ofstream soljson(argv[3]);
    soljson << solution;
    soljson.close();

    if (rank == 0)
    {
        //READ OUT JSONS
        
        timed *= 1.e3;

        double n_timesteps = tfm/dt;

        double per_ts = timed/n_timesteps;

        cout << n_timesteps << " timesteps" << endl;
        cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

        json timing;
        timing[dv][tpb][gpuA] = per_ts;

        std::ofstream timejson(argv[4]);
        timejson << std::setw(4) << timing << std::endl;
        timejson.close();
    }

    if (cGlob.hasGpu)
    {
        hipDeviceSynchronize();
        hipEventDestroy( start );
        hipEventDestroy( stop );
        for (int k=0; k<3; k++)
        {
            cudaHostFree(xpts[k]);
            cudaHostFree(state[k]);
        }
        
        delete[] xpts;
        delete[] state;
        hipDeviceReset();
    }
    else
    {
        cudaHostFree(xpts[0])
        cudaHostFree(state[0])
        delete[] xpts;
        delete[] states;
    }
	return 0;
}