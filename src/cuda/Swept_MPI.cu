#include "hip/hip_runtime.h"

// This file uses vector types to hold the dependent variables so fundamental operations on those types are defined as macros to accommodate different data types.  Also, keeping types consistent for common constants (0, 1, 2, etc) used in computation has an appreciable positive effect on performance.
#ifndef REAL
    #define REAL            float
    #define REALtwo         float2
    #define REALthree       float3
    #define SQUAREROOT(x)   sqrtf(x)

    #define ZERO            0.0f
    #define QUARTER         0.25f
    #define HALF            0.5f
    #define ONE             1.f
    #define TWO             2.f
#else

    #define ZERO            0.0
    #define QUARTER         0.25
    #define HALF            0.5
    #define ONE             1.0
    #define TWO             2.0
    #define SQUAREROOT(x)   sqrt(x)
#endif



/**
    Builds an upright triangle using the swept rule.

    Upright triangle using the swept rule.  This function is called first using the initial conditions or after results are read out using downTriange.  In the latter case, it takes the result of down triangle as IC.

    @param IC Array of initial condition values in order of spatial point.
    @param outRight Array to store the right sides of the triangles to be passed.
    @param outLeft Array to store the left sides of the triangles to be passed.
*/
__global__
void
upTriangle(const REALthree *IC, REALthree *outRight, REALthree *outLeft)
{
	extern __shared__ REALthree temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tididx = threadIdx.x + 2; //Block Thread ID
    int tidxTop = tididx + dimens.base; //
    int k=4;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tididx] = IC[gid];

    __syncthreads();

	if (threadIdx.x > 1 && threadIdx.x <(blockDim.x-2))
	{
		temper[tidxTop] = eulerStutterStep(temper, tididx, false, false);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	while (k < (blockDim.x>>1))
	{
		if (threadIdx.x < (blockDim.x-k) && threadIdx.x >= k)
		{
            temper[tididx] += eulerFinalStep(temper, tidxTop, false, false);
		}

        k+=2;
		__syncthreads();

		if (threadIdx.x < (blockDim.x-k) && threadIdx.x >= k)
		{
            temper[tidxTop] = eulerStutterStep(temper, tididx, false, false);
		}

		k+=2;
		__syncthreads();

	}
    // Passes right and keeps left
    writeOutRight(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
}

/**
    Builds an inverted triangle using the swept rule.

    Inverted triangle using the swept rule.  downTriangle is only called at the end when data is passed left.  It's never split.  Sides have already been passed between nodes, but will be swapped and parsed by readIn function.

    @param IC Full solution at some timestep.
    @param inRight Array of right edges seeding solution vector.
    @param inLeft Array of left edges seeding solution vector.
*/
__global__
void
downTriangle(REALthree *IC, const REALthree *inRight, const REALthree *inLeft)
{
	extern __shared__ REALthree temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tididx = threadIdx.x + 2;
    int tidxTop = tididx + dimens.base;
    int k = dimens.hts[2];

	readIn(temper, inRight, inLeft, threadIdx.x, gid);

    const char4 truth = {gid == 0, gid == 1, gid == dimens.idxend_1, gid == dimens.idxend};

    __syncthreads();

	while(k>1)
	{
		if (tididx < (dimens.base-k) && tididx >= k)
		{
            temper[tidxTop] = eulerStutterStep(temper, tididx, truth.y, truth.z);
		}

        k-=2;
        __syncthreads();

        if (!truth.x && !truth.w && tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tididx] += eulerFinalStep(temper, tidxTop, truth.y, truth.z);

        }

		k-=2;
		__syncthreads();
	}

    IC[gid] = temper[tididx];
}


/**
    Builds an diamond using the swept rule after a left pass.

    Unsplit diamond using the swept rule.  wholeDiamond must apply boundary conditions only at it's center.

    @param inRight Array of right edges seeding solution vector.
    @param inLeft Array of left edges seeding solution vector.
    @param outRight Array to store the right sides of the triangles to be passed.
    @param outLeft Array to store the left sides of the triangles to be passed.
    @param Full True if there is not a node run on the CPU, false otherwise.
*/
__global__
void
wholeDiamond(const REALthree *inRight, const REALthree *inLeft, REALthree *outRight, REALthree *outLeft, const bool split)
{

    extern __shared__ REALthree temper[];

    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tididx = threadIdx.x + 2;
    int tidxTop = tididx + dimens.base;

    char4 truth = {gid == 0, gid == 1, gid == dimens.idxend_1, gid == dimens.idxend};

    if (split)
    {
        gid += blockDim.x;
        truth.x = false, truth.y = false, truth.z = false, truth.w = false;
    }

    readIn(temper, inRight, inLeft, threadIdx.x, gid);

    __syncthreads();

    int k = dimens.hts[0];

    if (tididx < (dimens.base-dimens.hts[2]) && tididx >= dimens.hts[2])
    {
        temper[tidxTop] = eulerStutterStep(temper, tididx, truth.y, truth.z);
    }

    __syncthreads();

    while(k>4)
    {
        if (tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tididx] += eulerFinalStep(temper, tidxTop, truth.y, truth.z);
        }

        k -= 2;
        __syncthreads();

        if (tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tidxTop] = eulerStutterStep(temper, tididx, truth.y, truth.z);
        }

        k -= 2;
        __syncthreads();
    }

    // -------------------TOP PART------------------------------------------

    if (!truth.w  &&  !truth.x)
    {
        temper[tididx] += eulerFinalStep(temper, tidxTop, truth.y, truth.z);
    }

    __syncthreads();

    if (tididx > 3 && tididx <(dimens.base-4))
	{
        temper[tidxTop] = eulerStutterStep(temper, tididx, truth.y, truth.z);
	}


    k=6;
	__syncthreads();

	while(k<dimens.hts[4])
	{
		if (tididx < (dimens.base-k) && tididx >= k)
		{
            temper[tididx] += eulerFinalStep(temper, tidxTop, truth.y, truth.z);
        }

        k+=2;
        __syncthreads();

        if (tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tidxTop] = eulerStutterStep(temper, tididx, truth.y, truth.z);
		}
		k+=2;
		__syncthreads();
	}

    if (split)
    {
        writeOutLeft(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
    }
    else
    {
        writeOutRight(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
    }
}


//Split one is always first.
__global__
void
splitDiamond(REALthree *inRight, REALthree *inLeft, REALthree *outRight, REALthree *outLeft)
{
    extern __shared__ REALthree temper[];

    //Same as upTriangle
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tididx = threadIdx.x + 2;
    int tidxTop = tididx + dimens.base;
    int k = dimens.hts[2];

	readIn(temper, inRight, inLeft, threadIdx.x, gid);

    const char4 truth = {gid == dimens.hts[0], gid == dimens.hts[1], gid == dimens.hts[2], gid == dimens.hts[3]};

    __syncthreads();

    if (truth.z)
    {
        temper[tididx] = dbd[0];
        temper[tidxTop] = dbd[0];
    }
    if (truth.y)
    {
        temper[tididx] = dbd[1];
        temper[tidxTop] = dbd[1];
    }

    __syncthreads();

    while(k>0)
    {

        if (!truth.y && !truth.z && tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tidxTop] = eulerStutterStep(temper, tididx, truth.w, truth.x);
        }

        k -= 2;
        __syncthreads();

        if (!truth.y && !truth.z && tididx < (dimens.base-k) && tididx >= k)
        {
            temper[tididx] += eulerFinalStep(temper, tidxTop, truth.w, truth.x);
        }

        k -= 2;
        __syncthreads();
    }

    if (!truth.y && !truth.z && threadIdx.x > 1 && threadIdx.x <(blockDim.x-2))
	{
        temper[tidxTop] = eulerStutterStep(temper, tididx, truth.w, truth.x);
	}

	__syncthreads();
    k=4;

    //The initial conditions are timslice 0 so start k at 1.
    while(k<dimens.hts[2])
    {
        if (!truth.y && !truth.z && threadIdx.x < (blockDim.x-k) && threadIdx.x >= k)
        {
            temper[tididx] += eulerFinalStep(temper, tidxTop, truth.w, truth.x);

        }

        k+=2;
        __syncthreads();

        if (!truth.y && !truth.z && threadIdx.x < (blockDim.x-k) && threadIdx.x >= k)
        {
            temper[tidxTop] = eulerStutterStep(temper, tididx, truth.w, truth.x);
        }
        k+=2;
        __syncthreads();

    }

	writeOutLeft(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
}


using namespace std;

__host__
__forceinline__
REAL
energy(REALthree subj)
{
    REAL u = subj.y/subj.x;
    return subj.z/subj.x - HALF*u*u;
}

// This  is part of the MPI routine now kinda.
__host__
void
CPU_diamond(REALthree *temper, int htcpu[5])
{

    omp_set_num_threads(8);

    temper[htcpu[2]] = bd[0];
    temper[htcpu[2]+dimz.base] = bd[0];

    temper[htcpu[1]] = bd[1];
    temper[htcpu[1]+dimz.base] = bd[1];

    //Splitting it is the whole point!
    for (int k = htcpu[0]; k>0; k-=4)
    {
        #pragma omp parallel for
        for(int n = k; n<(dimz.base-k); n++)
        {
            if (n!=htcpu[1] && n!=htcpu[2])
            {
                temper[n+dimz.base] = eulerStutterStep(temper, n, (n==htcpu[3]),(n==htcpu[0]));
            }
        }

        #pragma omp parallel for
        for(int n = k-2; n<(dimz.base-(k-2)); n++)
        {
            if (n!=htcpu[1] && n!=htcpu[2])
            {
                temper[n] += eulerFinalStep(temper, n+dimz.base, n==htcpu[3],(n==htcpu[0]));
            }
        }
    }

    #pragma omp parallel for
    for(int n = 4; n < (dimz.base-4); n++)
    {
        if (n!=htcpu[1] && n!=htcpu[2])
        {
            temper[n+dimz.base] = eulerStutterStep(temper, n, (n==htcpu[3]),(n==htcpu[0]));
        }
    }

    //Top part.
    for (int k = 6; k<htcpu[2]; k+=4)
    {
        #pragma omp parallel
        for(int n = k; n<(dimz.base-k); n++)
        {
            if (n!=htcpu[1] && n!=htcpu[2])
            {
                temper[n] += eulerFinalStep(temper, n + dimz.base, (n==htcpu[3]), (n==htcpu[0]));
            }
        }

        #pragma omp parallel for
        for(int n = (k+2); n<(dimz.base-(k+2)); n++)
        {
            if (n!=htcpu[1] && n!=htcpu[2])
            {
                temper[n+dimz.base] = eulerStutterStep(temper, n, (n==htcpu[3]),(n==htcpu[0]));
            }
        }
    }
}

//The wrapper that calls the routine functions.
double
sweptWrapper(const int bks, int tpb, const int dv, const double dt, const double t_end, const int cpu,
    REALthree *IC, REALthree *T_f, const double freq, ofstream &fwr)
{
    const size_t smem = (2*dimz.base)*sizeof(REALthree);
    const int cpuLoc = dv-tpb;

    int htcpu[5];
    for (int k=0; k<5; k++) htcpu[k] = dimz.hts[k]+2;

	REALthree *d_IC, *d0_right, *d0_left, *d2_right, *d2_left;

	hipMalloc((void **)&d_IC, sizeof(REALthree)*dv);
	hipMalloc((void **)&d0_right, sizeof(REALthree)*dv);
	hipMalloc((void **)&d0_left, sizeof(REALthree)*dv);
    hipMalloc((void **)&d2_right, sizeof(REALthree)*dv);
	hipMalloc((void **)&d2_left, sizeof(REALthree)*dv);

	hipMemcpy(d_IC,IC,sizeof(REALthree)*dv,hipMemcpyHostToDevice);

	// Start the counter and start the clock.
	const double t_fullstep = 0.25*dt*(double)tpb;

	upTriangle <<<bks, tpb, smem>>> (d_IC, d0_right, d0_left);

    double t_eq;
    double twrite = freq - QUARTER*dt;

	// Call the kernels until you reach the final time

    if (cpu)
    {
        cout << "Hybrid Swept scheme" << endl;

        REALthree *h_right, *h_left;
        REALthree *tmpr = (REALthree *) malloc(smem);
        hipHostAlloc((void **) &h_right, tpb*sizeof(REALthree), hipHostMallocDefault);
        hipHostAlloc((void **) &h_left, tpb*sizeof(REALthree), hipHostMallocDefault);

        t_eq = t_fullstep;

        hipStream_t st1, st2, st3;
        hipStreamCreate(&st1);
        hipStreamCreate(&st2);
        hipStreamCreate(&st3);

        //Split Diamond Begin------

        wholeDiamond <<<bks-1, tpb, smem, st1>>> (d0_right, d0_left, d2_right, d2_left, true);

        hipMemcpyAsync(h_left, d0_left, tpb*sizeof(REALthree), hipMemcpyDeviceToHost, st2);
        hipMemcpyAsync(h_right, d0_right, tpb*sizeof(REALthree), hipMemcpyDeviceToHost, st3);

        hipStreamSynchronize(st2);
        hipStreamSynchronize(st3);

        // CPU Part Start -----

        for (int k=0; k<tpb; k++)  readIn(tmpr, h_right, h_left, k, k);

        CPU_diamond(tmpr, htcpu);

        for (int k=0; k<tpb; k++)  writeOutLeft(tmpr, h_right, h_left, k, k, 0);

        hipMemcpyAsync(d2_right, h_right, tpb*sizeof(REALthree), hipMemcpyHostToDevice, st2);
        hipMemcpyAsync(d2_left + cpuLoc, h_left, tpb*sizeof(REALthree), hipMemcpyHostToDevice, st3);

        // CPU Part End -----

        while(t_eq < t_end)
        {
            wholeDiamond <<<bks, tpb, smem>>> (d2_right, d2_left, d0_right, d0_left, false);

            //Split Diamond Begin------

            wholeDiamond <<<bks-1, tpb, smem, st1>>> (d0_right, d0_left, d2_right, d2_left, true);

            hipMemcpyAsync(h_left, d0_left, tpb*sizeof(REALthree), hipMemcpyDeviceToHost, st2);
            hipMemcpyAsync(h_right, d0_right, tpb*sizeof(REALthree), hipMemcpyDeviceToHost, st3);

            hipStreamSynchronize(st2);
            hipStreamSynchronize(st3);

            // CPU Part Start -----

            for (int k=0; k<tpb; k++)  readIn(tmpr, h_right, h_left, k, k);

            CPU_diamond(tmpr, htcpu);

            for (int k=0; k<tpb; k++)  writeOutLeft(tmpr, h_right, h_left, k, k, 0);

            hipMemcpyAsync(d2_right, h_right, tpb*sizeof(REALthree), hipMemcpyHostToDevice, st2);
            hipMemcpyAsync(d2_left + cpuLoc, h_left, tpb*sizeof(REALthree), hipMemcpyHostToDevice, st3);

            // CPU Part End -----

            // Automatic synchronization with memcpy in default stream

            //Split Diamond End------

            t_eq += t_fullstep;

    	    if (t_eq > twrite)
    		{
    			downTriangle <<<bks, tpb, smem>>> (d_IC, d2_right, d2_left);

    			hipMemcpy(T_f, d_IC, sizeof(REALthree)*dv, hipMemcpyDeviceToHost);

                fwr << "Density " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << T_f[k].x << " ";
                fwr << endl;

                fwr << "Velocity " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << (T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << "Energy " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << energy(T_f[k]) << " ";
                fwr << endl;

                fwr << "Pressure " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << pressure(T_f[k]) << " ";
                fwr << endl;

                upTriangle <<<bks, tpb, smem>>> (d_IC, d0_right, d0_left);

    			splitDiamond <<<bks, tpb, smem>>> (d0_right, d0_left, d2_right, d2_left);

                t_eq += t_fullstep;

                twrite += freq;
    		}
        }

        hipHostFree(h_right);
        hipHostFree(h_left);
        hipStreamDestroy(st1);
        hipStreamDestroy(st2);
        hipStreamDestroy(st3);
        free(tmpr);

	}
    else
    {
        cout << "GPU only Swept scheme" << endl;
        splitDiamond <<<bks, tpb, smem>>> (d0_right, d0_left, d2_right, d2_left);
        t_eq = t_fullstep;

        while(t_eq < t_end)
        {

            wholeDiamond <<<bks, tpb, smem>>> (d2_right, d2_left, d0_right, d0_left, false);

            splitDiamond <<<bks, tpb, smem>>> (d0_right, d0_left, d2_right, d2_left);
            //So it always ends on a left pass since the down triangle is a right pass.
            t_eq += t_fullstep;

            if (t_eq > twrite)
    		{
    			downTriangle <<<bks, tpb, smem>>> (d_IC, d2_right, d2_left);

    			hipMemcpy(T_f, d_IC, sizeof(REALthree)*dv, hipMemcpyDeviceToHost);

                fwr << "Density " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << T_f[k].x << " ";
                fwr << endl;

                fwr << "Velocity " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << (T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << "Energy " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << energy(T_f[k]) << " ";
                fwr << endl;

                fwr << "Pressure " << t_eq << " ";
                for (int k = 1; k<(dv-1); k++) fwr << pressure(T_f[k]) << " ";
                fwr << endl;

    			upTriangle <<<bks, tpb, smem>>> (d_IC, d0_right, d0_left);

    			splitDiamond <<<bks, tpb, smem>>> (d0_right, d0_left, d2_right, d2_left);

                t_eq += t_fullstep;

    			twrite += freq;
    		}
        }
    }

    downTriangle <<<bks, tpb, smem>>> (d_IC, d2_right, d2_left);

	hipMemcpy(T_f, d_IC, sizeof(REALthree)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d0_right);
	hipFree(d0_left);
    hipFree(d2_right);
	hipFree(d2_left);

    return t_eq;
}


