#include "hip/hip_runtime.h"

#include <fstream>
#include "euler.h"
#include "decomp.h"
#include "classic.h"
#include "swept.h"
#include <iomanip>

/**
----------------------
    MAIN PART
----------------------
*/

#ifndef HDW
    #define HDW     "WORKSTATION.json"
#endif

using std::cout, std::endl, std::vector, std::string, std::ifstream, std::ostream;

vector<int> jsonP(jsons jp, size_t sz)
{
	vector <int> outv;
	for(int i=0; i<sz; i++)
	{
		outv.push_back(jp[i].asInt());
	}
	return outv;
}

int main(int argc, char *argv[])
{   
    makeMPI(argc, argv);

    string ext = ".json";
    string myrank = std::to_string(ranks[1]);
    string sout = argv[3];
    sout.append(myrank);
    sout.append(ext); 
    string scheme = argv[1];

    ifstream hwjson(HDW, ifstream::in);
    jsons hwJ;
    hwjson >> hwJ;
    hwjson.close();

    vector<int> gpuvec = jsonP(hwJ["GPU"], 1);
    vector<int> smGpu(gpuvec.size());
    vector<int> threadv =  jsonP(hwJ["nThreads"], 1);
    cGlob.nThreads=threadv[ranks[1]]; // Potetntial for non constant
    cGlob.hasGpu = gpuvec[ranks[1]];
    std::partial_sum(gpuvec.begin(), gpuvec.end(), smGpu.begin());
    cGlob.nGpu = smGpu.back();
    smGpu.insert(smGpu.begin(), 0);
    vector <int> myGPU = jsonP(hwJ["gpuID"], 1);
    int gpuID = myGPU[ranks[1]];
    
    // Equation, grid, affinity data
    ifstream injson(argv[2], ifstream::in);
    injson >> inJ;
    injson.close();

    parseArgs(argc, argv);
    initArgs();

    /*  
        Essentially it should associate some unique (UUID?) for the GPU with the CPU. 
        Pretend you now have a (rank, gpu) map in all memory. because you could just retrieve it with a function.
    */

    int exSpace = (scheme.compare("S")) ? cGlob.htp : 2;
    int strt = cGlob.xcpu * ranks[1] + cGlob.xg * cGlob.hasGpu * smGpu[ranks[1]]; 
    states *state;
    vector <int> xpts;
    xpts.push_back(strt-1) // Basic vector will have all pass, start, and split points.
    //The pattern is important. 

    int mon;

    if (cGlob.hasGpu)
    {
        //GPU set up. Which device, what precision, copy constants to device.
        hipSetDevice(gpuID);
        if (sizeof(REAL)>6) 
        {
            hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        }
        hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), &heqConsts, sizeof(eqConsts))

        // Add the other half of the CPU and the GPU alloc.
        xalloc = exSpace + cGlob.xg + cGlob.xcpu;
        hipMallocManaged((void **) &state, xalloc * cGlob.szState);

        for (int k=0; k <= xalloc; k++)  initialState(inJ, k + strt, state);
        for (int k=1; k<=xalloc; k++) solutionOutput(state, 0.0, xpts);                  
    }
    else 
    {
        xalloc = exSpace + cGlob.xcpu;
        malloc((void **) &state, xalloc * cGlob.szState);
        for (int k=0; k <= xalloc; k++)  initialState(inJ, k + strt, state);  
        for (int k=1; k<=xalloc-cGlob.ht; k++) solutionOutput(state, 0.0, xpts);
    }

    int tstep = 1;
    // Start the counter and start the clock.  Maybe should time it with MPI.  Still use cudaSynchronize for GPU nodes.
    MPI_Barrier(MPI_COMM_WORLD);
    hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
    hipEventRecord( start, 0);

    // Call the correct function with the correct algorithm.
    double tfm;

    if (!scheme.compare("C"))
    {
        tfm = classicWrapper(state, xpts, &tstep);
    }
    else if  (!scheme.compare("S"))
    {
       tfm = sweptWrapper(state, xpts, &tstep);
    }
    else
    {
        cerr << "Incorrect or no scheme given" << endl;
    }

    MPI_Barrier(MPI_COMM_WORLD);

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timed, start, stop);

    endMPI();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    int nomar;
    ofstream soljson(argv[3]);
    soljson << solution;
    soljson.close();

    if (!ranks[1])
    {
        //READ OUT JSONS
        
        timed *= 1.e3;

        double n_timesteps = tfm/cGlob.dt;

        double per_ts = timed/n_timesteps;

        cout << n_timesteps << " timesteps" << endl;
        cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

        // Equation, grid, affinity data
        try {
            ifstream tjson(argv[4], ifstream::in);
            tjson >> timing;
            tjson.close();
        }
        catch (...) {}

        string tpbs = std::to_string(cGlob.tpb);
        string nXs = std::to_string(cGlob.nX);
        string gpuAs = std::to_string(cGlob.gpuA);
        cout << cGlob.gpuA << endl;

        ofstream timejson(argv[4], ofstream::trunc);
        timing[tpbs][nXs][gpuAs] = per_ts;
        timejson << timing;
        timejson.close();
    }

    if (cGlob.hasGpu)
    {
        hipDeviceSynchronize();
        hipEventDestroy( start );
        hipEventDestroy( stop );
        hipFree(state);
        hipDeviceReset();
    }
    else
    {
        free(state);
    }
	return 0;
}
