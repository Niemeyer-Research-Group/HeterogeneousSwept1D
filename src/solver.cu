#include "hip/hip_runtime.h"
/**
    This file evaluates the Euler equations applied to the 1D Sod Shock Tube problem.  It demonstrates the numerical solution to this problem in parallel using the GPU. The solution procedure uses a second order finite volume scheme with a minmod limiter parameterized by the Pressure ratio at cells on a three point stencil.  The solution also uses a second-order in time (RK2 or midpoint) scheme.
    
    The problem may be evaluated in three ways: Classic, SharedGPU, and Hybrid.  Classic simply steps forward in time and calls the kernel once every timestep (predictor step or full step).  SharedGPU uses the GPU for all computation and applies the swept rule.  Hybrid applies the swept rule but computes the node on the boundary with the CPU.  
*/
/* 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

// Two primary strategies used in this code: global variables and templating using structures.

#include "decomposition/classicCore.h"
// #include "decomposition/sweptCore.h"

void exitMerge()
{
    system("json-merge path/to/jsons/*.json")
}

int main(int argc, char *argv[])
{   
    makeMPI(argc, &argv);

    if (!rank[1]) atexit(exitMerge);

    getDeviceInformation();

    states *state;
    double *xpts;

    std::ifstream injson(argv[1]);
    json inJ;
    injson >> inJ;
    injson.close();

    parseArgs(inJ, argc, &argv);
    initArgs(inJ);

    hBound[0] = {};
    hBound[1] = {};
    
    delegateDomain(double *xpts, states *state);

    for (int k=0; k<dv; k++) initialState(inJ, &state[k]->Q[0]);

    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const double dt = atof(argv[3]);
	const double tf = atof(argv[4]) - QUARTER*dt; //Finish time
    const double freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //2 for Alternate, 1 for GPUShared, 0 for Classic
    const int bks = dv/tpb; //The number of blocks
    const double dx = lx/((REAL)dv-TWO);
    char const *prec;
    prec = (sizeof(REAL)<6) ? "Single": "Double";

    eCheckIn(dv, tpb, argc); //Initial error checking.

    // We always know that there's some eqConsts struct that we need to 
    // to put into constant memory.
    // PROBABLY NEED TO CHECK TO MAKE SURE THERE"S A GPU FIRST.
    if (gpuYes) hipMemcpyToSymbol(HIP_SYMBOL(deqConsts),&heqConsts,sizeof(eqConsts));

    // Start the counter and start the clock.
    MPI_Barrier(MPI_COMM_WORLD);
    hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the correct function with the correct algorithm.
    cout << scheme << " " ;
    double tfm;
    if (scheme)
    {
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, scheme-1, IC, T_final, freq, fwr);
    }
    else
    {
        tfm = classicWrapper(state, xpts);
    }

    endMPI();

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    std::ofstream soljson(argv[2]);
    soljson << solution;

    if (rank == 0)
    {
        //READ OUT JSONS
        
        timed *= 1.e3;

        double n_timesteps = tfm/dt;

        double per_ts = timed/n_timesteps;

        cout << n_timesteps << " timesteps" << endl;
        cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

        json timing;
        timing[dv][tpb][gpuA] = per_ts;

        std::ofstream timejson(argv[2]);
        timejson << timing;
    }

    if (xgpu)
    {
        hipDeviceSynchronize();

        hipEventDestroy( start );
        hipEventDestroy( stop );
        hipDeviceReset();
    }

	return 0;
}