#include "hip/hip_runtime.h"
/**
    The Classic Functions for the stencil operation
*/

#include "classicCore.h"

/** 
    Classic kernel for simple decomposition of spatial domain.

    @param States The working array result of the kernel call before last (or initial condition) used to calculate the RHS of the discretization
    @param finalstep Flag for whether this is the final (True) or predictor (False) step
*/
__global__ void classicStep(states *state, int ts)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x + 1; //Global Thread ID (one extra)

    stepUpdate(state, gid, ts)
}

void classicStepCPU(states *state, int numx)
{
    #pragma omp parallel for
    for (int k=1; k<numx; k++)
    {
        stepUpdate(state, k, tstep)
    }
}

void classicPassLeft(states *state, int idxend)
{
    MPI_Sendrecv(&state[1], szState, struct_type, ranks[0], TAGS(tstep), 
                                    &state[idxend-1], szState, struct_type, ranks[1], TAGS(tstep),
                                    MPI_COMM_WORLD, &status); 
}

void classicPassRight(states *state, int idxend)
{
    MPI_Sendrecv(&state[idxend], szState, struct_type, ranks[2],
                                TAGS(tstep), &state, szState, struct_type, ranks[0], TAGS(tstep),
                                MPI_COMM_WORLD, &status); 
}

void classicPass(states *state, int idxend)
{
    if (!ranks[1]) classicPassLeft(states *state, int idxend); 

    if (ranks[1] < lastproc) classicPassRight(states *state, int idxend);
}

//Classic Discretization wrapper.
double classicWrapper(states *state, double *xpts)
{
    cout << "Classic Decomposition" << endl;

    double t_eq = 0.0;
    int tstep = 1; //Starts at 1 (Initial condition is 0)
    double twrite = freq - QUARTER*dt;

    if (xgpu) // If there's no gpu assigned to the process this is 0.
    {
        int bks = xgpu/tpb;
        int gpui = xcpu/2;
        int gpuf = gpui + xgpu + 2; 
        int idxend = gpui + 1;

        states *state1 = &state[0];
        states *state2 = &state[gpuf];
        int gpubytes =  szState * (xgpu + 2));

    // Four streams for four transfers to and from cpu.
        hipStream_t st1, st2, st3, st4);
        hipStreamCreate(&st1);
        hipStreamCreate(&st2);
        hipStreamCreate(&st3);
        hipStreamCreate(&st4);

        states *dState, *hState;

        hipHostAlloc((void **)&hState, gpubytes);
        hipMalloc((void **)&dState, gpubytes);

        // Copy the initial conditions to the device array.
        hipMemcpy(dState, &state[gpui], gpubytes, hipMemcpyHostToDevice);

        while (t_eq < t_end)
        {
            classicStepCPU(state1, idxend);
            classicStepCPU(state2, idxend);

            classicDecomp <<< bks,tpb >>> (dState, tstep);

            // Host to device first.  Fills the 0 and end members of array
            hipMemcpyAsync(dState, &state1, szState, hipMemcpyHostToDevice, st1);
            hipMemcpyAsync(dState, state2, szState, hipMemcpyHostToDevice, st2);
            hipMemcpyAsync(state1, dState, szState, hipMemcpyDeviceToHost, st3);
            hipMemcpyAsync(state2, dState, szState, hipMemcpyDeviceToHost, st4);

            classicPassRight(state2, idxend);
            classicPassLeft(state1, idxend);
            
            // Increment Counter and timestep
            if (MODULA(tstep)) t_eq += dt;
            tstep++

            if (t_eq > twrite)
            {
                hipMemcpy(&hState, dState, gpubytes, hipMemcpyDeviceToHost);

                #pragma omp parallel for
                for (int k=1; k<idxend; k++) solutionOutput(state1[k]->Q[0], xpts[k], t_eq);
                
                #pragma omp parallel for
                for (int k=1; k<idxend; k++) solutionOutput(state2[k]->Q[0], xpts[k+gpuf], t_eq);

                #pragma omp parallel for
                for (int k=1; k<xgpu; k++) solutionOutput(hState[k]->Q[0], xpts[k+gpui], t_eq);

                twrite += freq;
            }
        }

        hipMemcpy(&hState[gpui], dState, gpubytes, hipMemcpyDeviceToHost);

        hipStreamDestroy(st1);
        hipStreamDestroy(st2);
        hipStreamDestroy(st3);
        hipStreamDestroy(st4);

        #pragma omp parallel for
        for (int k=1; k<idxend; k++) solutionOutput(state1[k]->Q[0], xpts[k], t_eq);
        
        #pragma omp parallel for
        for (int k=1; k<idxend; k++) solutionOutput(state2[k]->Q[0], xpts[k+gpuf], t_eq);

        #pragma omp parallel for
        for (int k=1; k<xgpu; k++) solutionOutput(hState[k]->Q[0], xpts[k+gpui], t_eq);

        hipFree(dState);
        hipHostFree(hState);

    }
    else
    {

        while (t_eq < t_end)
        {

            classicStepCPU(state, xcpu + 1);
            if (MODULA(tstep)) t_eq += dt;
            tstep++
            classicPass(state, xcpu + 1);

            if (t_eq > twrite)
            {

                #pragma omp parallel for
                for (int k=1; k<xcpu+1; k++) solutionOutput(state[k]->Q[0], xpts[k], t_eq);

                twrite += freq;
            }
            
        }
        #pragma omp parallel for
        for (int k=1; k<xcpu+1; k++) solutionOutput(state[k]->Q[0], xpts[k], t_eq);
    }
    return t_eq;
}

